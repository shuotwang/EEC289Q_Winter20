#include "hip/hip_runtime.h"
//
//  main.cpp
//  FractionalCascading
//
//  Created by Vincent on 2020/3/2.
//  Copyright © 2020 Vincent. All rights reserved.
//

#include <array>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <iterator>
#include <hiprand.h>
#include "hiprand/hiprand_kernel.h"
#include <time.h>

using namespace std;

bool compare(int i, int j) {return (i<j);}

__host__
vector<int> GenerateDiffNumber(int min,int max,int num)
{
    int rnd;
    vector<int> diff;
    vector<int> tmp;

    for(int i = min;i < max+1 ; i++ )
    {
        tmp.push_back(i);
    }
    srand((unsigned)time(0));
    for(int i = 0 ; i < num ; i++)
    {
        do{
            rnd = min+rand()%(max-min+1);
     
        }while(tmp.at(rnd-min)==-1);
        diff.push_back(rnd);
        tmp.at(rnd-min) = -1;
    }
    return diff;
}

__host__
int** generateData(int row, int col, int maxValue, int* eachLen){
    // generate sorted data
    
    srand((unsigned) time(NULL));
    
    int** data = 0;
    data = new int* [row];
    
    vector<int> totalRandons = GenerateDiffNumber(1, 500, 200);
    int k = 0;

    for (int i = 0; i < row; i++){
        int actualLen = rand() % (col) + 1;//length of nonzero elements in each row
        eachLen[i] = actualLen;
        
        data[i] = new int[col];

        
         for (int j = 0; j < actualLen; j++){
           // data[i][j] = rand() % (maxValue) + 1;
           data[i][j] = totalRandons[k++];
        }
            
        if (actualLen > 1) std::sort(data[i], data[i] + actualLen, compare);
        
        for (int j = actualLen; j < col; j++){
            data[i][j] = -1;
        }
    }
    
    return data;
}

typedef struct {
    int value = -1;
    int thisPos = 0;
    int nextPos = 0;
}item;

__device__
void binarySearchGPU(int* arr, int arraySize, int key, int* &pos) {

	printf("here bsn\n");

	int l = 0;
    int r = arraySize - 1;
    int mid = (l+r)/2;
    
    while(l <= r){
        mid = (l + r)/2;
        if(arr[mid] == key){
            *pos = mid;
			break;
        }
        
        if(arr[mid] > key){
            r = mid - 1;
        }else {
            l = mid + 1;
        }
    }
    
    *pos = l;
}

__host__ __device__
int binarySearchItem(item* items, int arraySize, int key){
    
    int l = 0;
    int r = arraySize - 1;
    int mid = (l+r)/2;
    
    while(l <= r){
        mid = (l + r)/2;
        if(items[mid].value == key){
            return mid;
        }
        
        if(items[mid].value > key){
            r = mid - 1;
        }else {
            l = mid + 1;
        }
    }
    
    return l;
}


__host__ __device__
int binarySearchArr(int* arr, int arraySize, int key) {

    int l = 0;
    int r = arraySize - 1;

    while(l <= r) {
        int mid = (l+r)/2;
        if (arr[mid] == key){
            return mid;
        }

        if (arr[mid] > key) {
            r = mid - 1;
        }else {
            l = mid + 1;
        }
    }

    return l;
}

__host__
int findMaxLen(int numArray, int* eachLen, int* &newLen){

    if (numArray == 0) return 0;
    
    int maxValue = 0;

    int* eachLen1 = new int[numArray];
    for (int i = 0; i < numArray; i++){
        eachLen1[i] = eachLen[i];
    }

    // calculate the maxLen: iterate from bottom back to top
    // this len = thisLen + preLen/2
    for (int i = numArray - 2; i >= 0; i--){
        eachLen1[i] = eachLen[i] + eachLen1[i+1] / 2;
        if (eachLen1[i] > maxValue) maxValue = eachLen1[i];
        // printf("max%d len%d each%d each+%d \n", maxValue, eachLen1[i], eachLen[i], eachLen1[i+1]);
    }

    newLen = eachLen1;
    
//    for (int i = 0; i < numArray; i++){
//        printf("newLen%d ", newLen[i]);
//    }
//    printf("\n");
    
    return maxValue;
}

__host__
int findIdxInArray(int key, int size, int* array){
    
    int pivot, left = 0, right = size - 1;
    while (left <= right) {
      pivot = left + (right - left) / 2;
      if (array[pivot] == key) return pivot;
      if (key < array[pivot]) right = pivot - 1;
      else left = pivot + 1;
    }
    return left;
}

__host__
int findIdxInItem(int key, int size, item* items){
    int pivot, left = 0, right = size - 1;
    while (left <= right) {
      pivot = left + (right - left) / 2;
      if (items[pivot].value == key) return pivot;
      if (key < items[pivot].value) right = pivot - 1;
      else left = pivot + 1;
    }
    return left;
}

__host__
item** fractionalCascading(int** data, int numArray, int* eachLen, int* maxLength, int* &newLen){
    
    int maxLen = findMaxLen(numArray, eachLen, newLen);
    maxLength[0] = maxLen;
    item** items = 0;
    items = new item* [numArray];
    
    // initialize the last row
    int lastLen = eachLen[numArray - 1];
    items[numArray - 1] = new item[maxLen];
    for (int i = 0; i < maxLen; i++){
        if (i < lastLen){
            items[numArray - 1][i].value = data[numArray - 1][i];
            items[numArray - 1][i].thisPos = i;
            items[numArray - 1][i].nextPos = 0;
        }
    }
    

    // other rows, merge the next with this one
    for (int i = (numArray - 2); i >= 0; i--){
        
        // this line i, next line i + 1
        
        items[i] = new item[maxLen];
        int thisLen = eachLen[i];
        int nextLen = newLen[i + 1];
        int thisNewLen = thisLen + nextLen / 2;
        
        // x for this line, y for next line, z for new item array
        // place element first, then find positions
        int x = 0, y = 1, z = 0;
        while(x < thisLen && y < nextLen){
            if (z < thisNewLen) {
                if (data[i][x] <= items[i + 1][y].value){
                    items[i][z].value = data[i][x];
                    items[i][z].thisPos = x;
                    items[i][z].nextPos = binarySearchItem(items[i + 1], newLen[i + 1], data[i][x]);
                    x++;
                    z++;
                }else {
                    items[i][z].value = items[i + 1][y].value;
                    items[i][z].thisPos = binarySearchArr(data[i], thisLen, items[i + 1][y].value);
                    items[i][z].nextPos = y;
                    y += 2;
                    z++;
                }
            }
        }
        
        while (x < thisLen) {
            items[i][z].value = data[i][x];
            items[i][z].thisPos = x;
            items[i][z].nextPos = findIdxInItem(data[i][x], nextLen, items[i+1]);
            x++;
            z++;
        }
        
        while (y < nextLen) {
            items[i][z].value = items[i + 1][y].value;
            items[i][z].thisPos = findIdxInArray(items[i + 1][y].value, thisLen, data[i]);
            items[i][z].nextPos = y;
            y += 2;
            z++;
        }
    }
    
    return items;
}

__host__
void show2dData(int** data, int numArray, int maxArrayLen){
    printf("Sorted Raw Data:\n");
    for (int i = 0; i < numArray; i++){
        for (int j = 0; j < maxArrayLen; j++){
            printf("%d\t", data[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}


const int numArray = 10;
const int maxArrayLen = 5;
const int maxValue = 100;
const int length = 15;


__global__
void kernelSetRandom(hiprandState *curandStates, int N, long clock_for_rand){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx > 1000) return;
	// int stride = blockDim.x * gridDim.x;
	
	// if (idx > num)	return;
	for (int a = 0; a < 1000; a ++){
		hiprand_init(clock_for_rand, idx, 0, &curandStates[a]);
	}
}

__global__
void findPos(int** valueArr, int** thisPosArr, int** nextPosArr, int** data, int numArray, int* newLen, int* oriLen, int* valueArray, int N, int* keyOut, hiprandState* curand_states){
    
    // find idx of first row
	// int idx = binarySearchArr(valueArr[0], newLen[0], key);
	/*
	if (threadIdx.x==0){
		for(int i = 0; i < numArray; i++){
			for(int j = 0; j < length; j++){
				printf("%d ", valueArr[i][j]);
			}
		}
	}
	*/

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for (int a = threadIndex; a < N; a += stride){
		int b = a % 1000;
	// if (threadIndex > N)	return;
	hiprandState localState = curand_states[b];
	int key = (int)abs(hiprand(&localState)) % maxValue + 1;
	// printf("inside key: %d\n", key);
	*keyOut = key;
	
	int idx;
    int l = 0;
    int r = newLen[0] - 1;

    while(l <= r) {
        int mid = (l+r)/2;
        if (valueArr[0][mid] == key){
            idx = mid;
        }

        if (valueArr[0][mid] > key) {
            r = mid - 1;
        }else {
            l = mid + 1; 
        }
    }

    idx = l;
	// end of binary search	
	
    if (idx > newLen[0] - 1){
        idx = newLen[0] - 1;
    }
    
    int thisIdx = thisPosArr[0][idx];
    if (thisIdx > oriLen[0] - 1){
        thisIdx = oriLen[0] - 1;
    }
    
	valueArray[0] = data[0][thisIdx];
	//printf("thisIdx %d ", thisIdx); 

    int nextIdx = nextPosArr[0][idx];
    if (nextIdx > newLen[1] - 1) {
        nextIdx = newLen[1] - 1;
    }
	 

    // other lines
    for (int i = 1; i < numArray; i++){

		int selectedItemThisPos = thisPosArr[i][nextIdx];
		int selectedItemNextPos = nextPosArr[i][nextIdx];
		// printf("i %d, nextIdx %d, value %d\n", i, nextIdx, selectedItemValue);
        
        if ((nextIdx > 0) && valueArr[i][nextIdx - 1] > key){
			selectedItemThisPos = thisPosArr[i][nextIdx - 1];
			selectedItemNextPos = nextPosArr[i][nextIdx - 1];
        }
        
        if ((nextIdx < newLen[i] - 1) && valueArr[i][nextIdx + 1] < key){
			selectedItemThisPos = thisPosArr[i][nextIdx + 1];
			selectedItemNextPos = nextPosArr[i][nextIdx + 1];
        }
        
        thisIdx = selectedItemThisPos;
        if (thisIdx > oriLen[i] - 1){
            thisIdx = oriLen[i] - 1;
        }
        
        valueArray[i] = data[i][thisIdx];
        
        nextIdx = selectedItemNextPos;
        if ((i < numArray - 1) && (nextIdx > newLen[i + 1] - 1)){
            nextIdx = newLen[i + 1] - 1;
        }   
    }
}


/*
	for (int i = 0; i < numArray; i++){
		printf("%d ", valueArray[i]);
	}
*/
}

int main(int argc, const char * argv[]) {
    // eachLen (oriLen)    
    int* eachLen = new int[numArray];
	int* eachLenK;
	hipMallocManaged(&eachLenK, numArray * sizeof(int));
    
	// data
    int** data = generateData(numArray, maxArrayLen, maxValue, eachLen);
	hipMemcpy(eachLenK, eachLen, numArray * sizeof(int), hipMemcpyHostToDevice);
    show2dData(data, numArray, maxArrayLen);

	int dataSize = numArray*maxArrayLen;
	int* dataK; hipMallocManaged(&dataK, dataSize * sizeof(int));
	std::copy(&data[0][0], &data[0][0] + dataSize, dataK);

	// new length
    int* newLen = new int[numArray];
	int* newLenK;
	hipMallocManaged(&newLenK, numArray * sizeof(int));

	// items
	item** items = 0;
    int* maxLength = new int[1];
    items = fractionalCascading(data, numArray, eachLen, maxLength, newLen);
	hipMemcpy(newLenK, newLen, numArray * sizeof(int), hipMemcpyHostToDevice);
    
    // print items on screen
    printf("Data after fractional cascading:\n");
    for (int i = 0; i < numArray; i++){
        for (int j = 0; j < *maxLength; j++){
            printf("%3d[%d, %d] ", items[i][j].value, items[i][j].thisPos, items[i][j].nextPos);
        }
        printf("\n");
    }
    
	// get key
    // int key = data[0][2] + 1;
	
	// convert items struct to arrays
	// const int length = *maxLength;

	int** valueArr = new int* [numArray];
	int** thisPosArr = new int* [numArray];
	int** nextPosArr = new int* [numArray];
	for (int i = 0; i < numArray; i++) {
		valueArr[i] = new int[length];
		thisPosArr[i] = new int[length];
		nextPosArr[i] = new int[length];
		for (int j = 0; j < length; j++) {
			if (j < *maxLength) {
				valueArr[i][j] = items[i][j].value;
				thisPosArr[i][j] = items[i][j].thisPos;
				nextPosArr[i][j] = items[i][j].nextPos;
			}else{
				valueArr[i][j] = -1;
				thisPosArr[i][j] = 0;
				nextPosArr[i][j] = 0;
			}
		}	
	}



	// convert items arrays into cuda format
	// testing for new mem allocation
	int* data_valueArr = (int *)malloc(sizeof(int) * length * numArray);
	int* data_thisPosArr = (int *)malloc(sizeof(int) * length * numArray);
	int* data_nextPosArr = (int *)malloc(sizeof(int) * length * numArray);

	for (int i = 0; i < numArray; i++){
		for(int j = 0; j < length; j++){
		data_valueArr[i * length + j] = valueArr[i][j];
		data_thisPosArr[i * length + j] = thisPosArr[i][j];
		data_nextPosArr[i * length + j] = nextPosArr[i][j]; 
		}
	}
	
	int* data_data = (int *)malloc(sizeof(int) * length * numArray);
	for (int i = 0; i < numArray; i++) {
		for (int j = 0; j < maxArrayLen; j++){
			data_data[i * maxArrayLen + j] = data[i][j];
		}
	}


	int** d_valueArr;
	int* d_data_valueArr;
	int** d_thisPosArr;
	int* d_data_thisPosArr;
	int** d_nextPosArr;
	int* d_data_nextPosArr;
	int** d_data;
	int* d_data_data;
	hipMalloc((void**)&d_valueArr, sizeof(int *) * numArray);
	hipMalloc((void**)&d_data_valueArr, sizeof(int) * numArray * length);

	hipMalloc((void**)&d_thisPosArr, sizeof(int *) * numArray);
	hipMalloc((void**)&d_data_thisPosArr, sizeof(int) * numArray * length);

	hipMalloc((void**)&d_nextPosArr, sizeof(int *) * numArray);
	hipMalloc((void**)&d_data_nextPosArr, sizeof(int) * numArray * length);

	hipMalloc((void**)&d_data, sizeof(int *) * numArray);
	hipMalloc((void**)&d_data_data, sizeof(int) * numArray * maxArrayLen);

	for (int i = 0; i < numArray; i++){
		valueArr[i] = d_data_valueArr + length * i;
		thisPosArr[i] = d_data_thisPosArr + length * i;
		nextPosArr[i] = d_data_nextPosArr + length * i;
		data[i] = d_data_data + maxArrayLen * i;
	}

	hipMemcpy(d_valueArr, valueArr, sizeof(int*) * numArray, hipMemcpyHostToDevice);
	hipMemcpy(d_data_valueArr, data_valueArr, sizeof(int) * numArray * length, hipMemcpyHostToDevice);
	hipMemcpy(d_thisPosArr, thisPosArr, sizeof(int*) * numArray, hipMemcpyHostToDevice);
	hipMemcpy(d_data_thisPosArr, data_thisPosArr, sizeof(int) * numArray * length, hipMemcpyHostToDevice);
	hipMemcpy(d_nextPosArr, nextPosArr, sizeof(int*) * numArray, hipMemcpyHostToDevice);
	hipMemcpy(d_data_nextPosArr, data_nextPosArr, sizeof(int) * numArray * length, hipMemcpyHostToDevice);
	hipMemcpy(d_data, data, sizeof(int*) * numArray, hipMemcpyHostToDevice);
	hipMemcpy(d_data_data, data_data, sizeof(int) * numArray * maxArrayLen, hipMemcpyHostToDevice);

	int N = 1000000;
	int blockSize = 1024;
    int numBlock = (N + blockSize - 1) / blockSize;

	int* resultK;
	hipMallocManaged(&resultK, numArray * sizeof(int));
	
	// prepare for kernel random number generation (key for fc)
	hiprandState* dev_states;
	hipMalloc((void**) &dev_states, sizeof(hiprandState) * 1000);
	long clock_for_rand = clock();
	kernelSetRandom<<<numBlock, blockSize>>>(dev_states, N, clock_for_rand);

	// kernel FC function
	int* d_key;
	hipMallocManaged(&d_key, sizeof(int));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime;
	hipEventRecord(start, 0);
	
    findPos<<<numBlock, blockSize>>>(d_valueArr, 
									 d_thisPosArr, 
								     d_nextPosArr, 
									 d_data, 
									 numArray, newLenK, eachLenK, resultK, N, d_key, dev_states);
  	

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time: %f ms\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	hipDeviceSynchronize();

    
	int* h_key;
	h_key = (int *)malloc(sizeof(int));
	hipMemcpy(h_key, d_key, sizeof(int), hipMemcpyDeviceToHost);
    printf("\nkey: %d\n", h_key[0]);
	
	int* hostResult = new int[numArray];
	hostResult = (int *)malloc(numArray * sizeof(int));
	hipMemcpy(hostResult, resultK, numArray * sizeof(int), hipMemcpyDeviceToHost);
    printf("Values: ");
    for (int i = 0; i < numArray; i++){
        printf("%d ", hostResult[i]);
    }

	printf("\n"); 

	hipFree(eachLen);
	hipFree(newLenK);
	hipFree(d_valueArr);
	hipFree(d_data_valueArr);
	hipFree(d_thisPosArr);
	hipFree(d_data_thisPosArr);
	hipFree(d_nextPosArr);
	hipFree(d_data_nextPosArr);
	hipFree(d_data);
	hipFree(d_data_data);

    
    return 0;
}
